
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

int main()
{
    printf("Hello World from CPU!\n");
    helloFromGPU<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
